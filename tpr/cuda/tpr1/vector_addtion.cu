#include "hip/hip_runtime.h"
#include "helper_timer.h"
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define out
#define null NULL;


__global__ void vector_add_kernel (int *a, int *b, int *c, int N) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;	
	if (tid < N) {
		c[tid] = a[tid] + b[tid];
	}
}

void gpu_vector_add(int *a, int *b, int *c, int N, 
		out float* kernel_time, out float* transfer_time) {

	int SIZE = N * sizeof(int);
	StopWatchInterface* transfer_timer = null;
	StopWatchInterface* kernel_timer = null;
	sdkCreateTimer(&transfer_timer);
	sdkCreateTimer(&kernel_timer);
	
	kernel_timer->reset();
	transfer_timer->reset();
	
	int *dev_a, *dev_b, *dev_c;

	transfer_timer->start();
	hipMalloc((void**)&dev_a, SIZE);
	hipMalloc((void**)&dev_b, SIZE);
	hipMalloc((void**)&dev_c, SIZE);
           
	hipMemcpy(dev_a, a, SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, SIZE, hipMemcpyHostToDevice);	
	
	
	const int BLOCK_SIZE = 1024;
	const int MAX_BLOCK_SIZE = 65536;
	int k = ceil(((double)N) / BLOCK_SIZE);
	
	
	dim3 thredsPerBlock(BLOCK_SIZE, 1, 1);
    dim3 numBlock(
			k <= MAX_BLOCK_SIZE ? k : MAX_BLOCK_SIZE, 
			ceil(((double)k) / MAX_BLOCK_SIZE),
			1); 
	transfer_timer->stop();
			
	kernel_timer->start();
	vector_add_kernel<<<numBlock, thredsPerBlock>>>(dev_a, dev_b, dev_c, N);
	hipDeviceSynchronize();
	kernel_timer->stop();
	
	transfer_timer->start();
	hipMemcpy(c, dev_c, SIZE, hipMemcpyDeviceToHost);
	
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	transfer_timer->stop();
	
	*kernel_time = kernel_timer->getTime();
	*transfer_time = transfer_timer->getTime();
	sdkDeleteTimer(&transfer_timer);
	sdkDeleteTimer(&kernel_timer);
}


void cpu_vector_add(int *a, int *b, int *c, int N, 
		out float* time) {
	StopWatchInterface* timer = null;
	sdkCreateTimer(&timer);
	timer->reset();
	timer->start();
	
	for ( int i = 0; i < N; ++i) {
		c[i] = a[i] + b[i];
	}
	timer->stop();
	*time = timer->getTime();
	sdkDeleteTimer(&timer);
}

bool compare(int *a, int *b, int N) {
	for (int i = 0; i < N; ++i) {
		if (a[i] != b[i]) {
			return a[i] - b[i];
		}
	}
	return 0;
}

int main(void) {
	const int TESTS = 100;
	int size;
	int *a, *b, *c_cpu, *c_gpu;
	float gpu_kernel, gpu_transfer, cpu;
	float t_gpu_kernel, t_gpu_transfer, t_cpu;
	
	int tests[] = {1, 
		2,		4,		8,		16,		32,		64,		128,	256,	512,	1024,
		2000,	4000,	8000,	16000,	32000,	64000,	128000,	256000,	512000,	1024000,
		2000000,4000000,8000000,16000000 };
	
	for (int k = 0; k < sizeof(tests) / sizeof (int); ++k) {
		size = tests[k];
	/*while (true) {
		std::cin >> size;
		
		if (size <= 0) {
			break;
		}*/
	
		a = new int[size];
		b = new int[size];
		c_cpu = new int[size];
		c_gpu = new int[size];
			
		for (int i = 0; i < size; ++i) {
			a[i] = i;
			b[i] = i * i;
		}

		gpu_kernel = gpu_transfer = cpu = 0.0f;
		for (int i = 0; i < TESTS; ++i) {
			gpu_vector_add(a, b, c_gpu, size, &t_gpu_kernel, &t_gpu_transfer);
			cpu_vector_add(a, b, c_cpu, size, &t_cpu);
			cpu += t_cpu;
			gpu_kernel += t_gpu_kernel;
			gpu_transfer += t_gpu_transfer;			
		}
		
		gpu_transfer /= TESTS;
		gpu_kernel /= TESTS;
		cpu /= TESTS;
		
		if (size == 1) continue;
		/*std::cout << (compare(c_cpu, c_gpu, size)? "  INVALID !!!: " : "valid: ") 
				<< "size: " << size
				<< " cpu_time: "	<< cpu << " [ms] gpu_kernel: " << gpu_kernel 
				<< "[ms] gpu_transfer: " << gpu_transfer << "[ms] gpu_all: "
				<< gpu_kernel + gpu_transfer << "[ms]\n";		*/
		std::cout << gpu_kernel << "\n";
	}
	
	return 0;
}