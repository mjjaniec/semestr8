#include "hip/hip_runtime.h"
/*** Calculating a derivative with CD ***/
#include "helper_timer.h"
#include <iostream>
#include <fstream>
#include <cmath>

float get_time() {
	static StopWatchInterface* timer = NULL;
	if (!timer) {
		sdkCreateTimer(&timer);
		timer->reset();
		timer->start();
	}
	return timer->getTime();
}

void checkErrors(char *label)
{
	// we need to synchronise first to catch errors due to
	// asynchroneous operations that would otherwise
	// potentially go unnoticed
	hipError_t err;
	err = hipDeviceSynchronize();
	if (err != hipSuccess)
	{
		char *e = (char*) hipGetErrorString(err);
		fprintf(stderr, "CUDA Error: %s (at %s)\n", e, label);
	}
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		char *e = (char*) hipGetErrorString(err);
		fprintf(stderr, "CUDA Error: %s (at %s)\n", e, label);
	}
}

__global__ void copy_array(float *u, float *u_prev, int N, int BSZ)
{
	int i = threadIdx.x;
	int j = threadIdx.y;
	int I = blockIdx.y*BSZ*N + blockIdx.x*BSZ + j*N + i;
	if (I>=N*N){return;}	
	u_prev[I] = u[I];
}

// GPU kernel
__global__ void update (float *u, float *u_prev, int N, float h, float dt, float alpha, int BSZ)
{
	// Setting up indices
	int i = threadIdx.x;
	int j = threadIdx.y;
	int I = blockIdx.y*BSZ*N + blockIdx.x*BSZ + j*N + i;
	
	if (I>=N*N)
	{
		return;
	}	
	//if (()>=N || j>){return;}	

	
	// if not boundary do
	if ( (I>N) && (I< N*N-1-N) && (I%N!=0) && (I%N!=N-1)) 
	{	
		u[I] = u_prev[I] + alpha*dt/(h*h) * (u_prev[I+1] + u_prev[I-1] + u_prev[I+N] + u_prev[I-N] - 4*u_prev[I]);
	}
	
	// Boundary conditions are automatically imposed
	// as we don't touch boundaries
}

int main()
{
	int tests[] = {4, 32, 64, 96, 128, 192, 256, 384, 512};
	std::cout<<"size overall kernel transport\n";
	for(int N : tests) {
		// Allocate in CPU
		int BLOCKSIZE = 16;

		float xmin 	= 0.0f;
		float xmax 	= 3.5f;
		float ymin 	= 0.0f;
		//float ymax 	= 2.0f;
		float h   	= (xmax-xmin)/(N-1);
		float dt	= 0.00001f;	
		float alpha	= 0.645f;
		float time 	= 0.4f;

		int steps = ceil(time/dt);
		int I;

		float *x  	= new float[N*N]; 
		float *y  	= new float[N*N]; 
		float *u  	= new float[N*N];
		float *u_prev	= new float[N*N];


		// Generate mesh and intial condition
		for (int j=0; j<N; j++)
		{	
			for (int i=0; i<N; i++)
			{	
				I = N*j + i;
				x[I] = xmin + h*i;
				y[I] = ymin + h*j;
				u[I] = 0.0f;
				if ( (i==0) || (j==0)) 
				{
					u[I] = 200.0f;
				}
			}
		}

		// Allocate in GPU
		float *u_d, *u_prev_d;
		float kernel_start, kernel_stop;
		
		double start = get_time();
		hipMalloc( (void**) &u_d, N*N*sizeof(float));
		hipMalloc( (void**) &u_prev_d, N*N*sizeof(float));

		// Copy to GPU
		hipMemcpy(u_d, u, N*N*sizeof(float), hipMemcpyHostToDevice);

		// Loop 
		dim3 dimGrid(int((N-0.5)/BLOCKSIZE)+1, int((N-0.5)/BLOCKSIZE)+1);
		dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
		kernel_start = get_time();
		for (int t=0; t<steps; t++)
		{	
			copy_array <<<dimGrid, dimBlock>>> (u_d, u_prev_d, N, BLOCKSIZE);
			update <<<dimGrid, dimBlock>>> (u_d, u_prev_d, N, h, dt, alpha, BLOCKSIZE);
		}
		kernel_stop = get_time();
		checkErrors("update");
		
				// Free device
		
		if (N < 10) continue;
		// Copy result back to host
		hipMemcpy(u, u_d, N*N*sizeof(float), hipMemcpyDeviceToHost);
		hipFree(u_d);
		hipFree(u_prev_d);
		
		double stop = get_time();
		
		
		double elapsed = stop - start;
		double kernel_elapsed = kernel_stop - kernel_start;
		std::cout<<N<<" "<<elapsed<<" "<<kernel_elapsed<<" "<<elapsed-kernel_elapsed<<std::endl;
	}
}
