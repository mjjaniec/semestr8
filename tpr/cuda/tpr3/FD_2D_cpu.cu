#include "hip/hip_runtime.h"
/*** Calculating a derivative with CD ***/
#include "helper_timer.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include <omp.h>

float get_time() {
	static StopWatchInterface* timer = NULL;
	if (!timer) {
		sdkCreateTimer(&timer);
		timer->reset();
		timer->start();
	}
	return timer->getTime();
}


void update (float *u, float *u_prev, int N, float h, float dt, float alpha, int BSZ)
{
	int MAX = N*N-1-N;
	#pragma omp parallel for
	for(int I = N+1; I < MAX; ++I) {
		if ( (I%N!=0) && (I%N!=N-1)) {	
			u[I] = u_prev[I] + alpha*dt/(h*h) * (u_prev[I+1] + u_prev[I-1] + u_prev[I+N] + u_prev[I-N] - 4*u_prev[I]);
		}
	}
	
	// Boundary conditions are automatically imposed
	// as we don't touch boundaries
}

int main()
{
	omp_set_num_threads(4);
	// Allocate in CPU
	int tests[] = {32, 64, 96, 128, 192, 256};
	for(int N : tests) {
		int BLOCKSIZE = 16;

		float xmin 	= 0.0f;
		float xmax 	= 3.5f;
		float ymin 	= 0.0f;
		//float ymax 	= 2.0f;
		float h   	= (xmax-xmin)/(N-1);
		float dt	= 0.00001f;	
		float alpha	= 0.645f;
		float time 	= 0.4f;

		int steps = ceil(time/dt);
		int I;

		float *x  	= new float[N*N]; 
		float *y  	= new float[N*N]; 
		float *u  	= new float[N*N];
		float *u_prev	= new float[N*N];


		// Generate mesh and intial condition
		for (int j=0; j<N; j++)
		{	
			for (int i=0; i<N; i++)
			{	
				I = N*j + i;
				x[I] = xmin + h*i;
				y[I] = ymin + h*j;
				u[I] = 0.0f;
				if ( (i==0) || (j==0)) 
				{
					u[I] = 200.0f;
				}
			}
		}

		
		double start = get_time();
		for (int t=0; t<steps; t++)
		{	
			memcpy(u_prev, u, N * sizeof(float));
			update(u, u_prev, N, h, dt, alpha, BLOCKSIZE);
		}
		double stop = get_time();
		
		double elapsed = stop - start;
		std::cout<<N<<" "<<elapsed<<std::endl;

	}
	
}
