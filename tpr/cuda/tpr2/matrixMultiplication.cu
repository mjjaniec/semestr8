#include "hip/hip_runtime.h"
// Matrix multiplication by parts
// Elements stored in row-major order

using namespace std;
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

typedef struct
{	int width;
	int height;
	float *elements;
} Matrix;

// Forward declaration of matrix mult
__global__ void MatMulKernel (const Matrix, const Matrix, Matrix);

// Host code
void GpuMatMul(const Matrix A, const Matrix B, Matrix C)
{
	// Load matrices A and B to device memory
	Matrix d_A;
	d_A.width = A.width; d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc((void**) &d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	
	Matrix d_B;
	d_B.width = B.width; d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc((void**) &d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	
	// allocate C in device
	Matrix d_C;
	d_C.width = C.width; d_C.height = C.height;
	size = d_C.width * d_C.height * sizeof(float);
	hipMalloc((void**) &d_C.elements, size);
	
	// call kernel
    dim3 thredsPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1); // threads per block?
    dim3 numBlock(ceil(((double)A.width)/BLOCK_SIZE), ceil(((double)A.width)/BLOCK_SIZE), 1); // number of blocks?
	MatMulKernel<<<numBlock, thredsPerBlock>>>(d_A, d_B, d_C);
	hipDeviceSynchronize();
	
	// copy C to host
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	
	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

//matrix multiplication kernel
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
	// each thread computes one element of C and acumulates results to Cvalue 
	float Cvalue = 0; 
	int row = blockIdx.y * blockDim.y + threadIdx.y; 
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if ((row>=A.height) || (col>=B.width)){
		return;
	}
	for (int e=0; e<A.width; e++) {
		Cvalue += A.elements[row*A.width + e] * B.elements[e*B.width + col];
	}
	C.elements[row*C.width + col] = Cvalue;
}

void CpuMatMul(const Matrix A, const Matrix B, Matrix C)
{
	float sum;
	int N = A.width;
	for (int row = 0; row < N; ++row)
	{
		for (int column = 0; column < N; ++column) 
		{
			sum = 0.0f;
			for (int k = 0; k < N; ++k) {
				sum += A.elements[row * N + k] * B.elements[k * N + column];
			}			
			C.elements[row * N + column] = sum;
		}
	}
}

int main(int argc, char ** argv)
{	
	int tests[] = {452};
	for (int test = 0; test < sizeof(tests)/sizeof(int); ++ test)
	{
		int Width = tests[test];
		
		Matrix A;
		Matrix B;
		Matrix C;
		
		A.width = Width;
		B.width = Width;
		C.width = Width;
		
		A.height = Width;
		B.height = Width;
		C.height = Width;
		
		A.elements = new float[Width*Width];
		B.elements = new float[Width*Width];
		C.elements = new float[Width*Width];
		
		//fill matrices
		std::ifstream A_input;
		std::ifstream B_input;
		
		float a, b;
		for (int i = 0, max = Width * Width; i < max; ++i)
		{		
			A.elements[i] = rand();
			B.elements[i] = rand();
		}
		A_input.close();
		B_input.close();
		
		const int N = 100;
		float gpuTime, cpuTime;
		hipEvent_t start, stop;
		
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);

		for (int i = 0; i < N; ++i) 
		{
			GpuMatMul(A, B, C);		
		}
		
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&gpuTime, start, stop);
		gpuTime /= N;
		
		
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
		
		/*for (int i = 0; i < N; ++i)
		{
			CpuMatMul(A, B, C);
		}*/
		
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&cpuTime, start, stop);
		cpuTime /= N;
		
		/*
		CpuMatMul(A, B, C);
		std::ofstream C_output;
		C_output.open("C.txt");
		for (int i=0; i<Width; i++)
		{	for (int j=0; j<Width; j++)
				C_output<<C.elements[i*Width+j]<<"\t";
			C_output<<endl;
		}
		*/
		
		cout << "matrixSize: " << Width << " gpuTime: " << gpuTime << "[ms] cpuTime: " << cpuTime << "[ms]\n";
	}
}
